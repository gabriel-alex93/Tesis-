#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <omp.h>
#include <math.h>
#include "hip/hip_fp16.h"


int PrintCu(const int N,int *a);
__global__ void MetNormal(int *a_d,const int N );
__global__ void diagonal(int *a_d ,const int N ,int Nm ,int xi ,int yi,int zi,int n_rec);
__global__ void rec(int *a_d ,const int N ,int Nm ,int xi ,int yi,int zi,int n_rec,const  int  N_CORTE, const int BlockSize){

	int x=blockIdx.x*blockDim.x+threadIdx.x+xi;
	int x0=blockIdx.x*blockDim.x+threadIdx.x;
	int y=blockIdx.y*blockDim.y+threadIdx.y+yi;
	int y0=blockIdx.y*blockDim.y+threadIdx.y;
	int z=blockIdx.z*blockDim.z+threadIdx.z+zi;
	int z0=blockIdx.z*blockDim.z+threadIdx.z;
	int tid1=z*N*N+y*N+x;
	int tid=z0*N*N+y0*N+x0;

	int NmF= (int)ceil((float)((Nm)/2.0f));
	//printf("pspace x y z (%i %i %i)   dominio x y z (%i %i %i)  tid1 dominio %i  tid pspace %i \n",x0, y0, z0, x, y, z,tid1,tid);
 
	//if(Nm>=BlockSize ){	
	if(x < N && y < N && z < N){
		//a_d[tid1]=n_rec;
		a_d[tid1]=1;
		if(x+y >z){
			a_d[tid1]=9;
		}
	}
	//}
	if(Nm<=N_CORTE && tid==0){ //  AQUI NCORTE(pensar en calcular N_corte en base al basico optimo <7>)

		int Nbloques= (Nm+BlockSize-1)/BlockSize;

		dim3 b(BlockSize,BlockSize,BlockSize);
		dim3 g(Nbloques,Nbloques,Nbloques);

		hipStream_t s1;
		hipStream_t s2;
		hipStream_t s3;

		hipStreamCreateWithFlags(&s1,hipStreamNonBlocking);
		hipStreamCreateWithFlags(&s2,hipStreamNonBlocking);		      
		hipStreamCreateWithFlags(&s3,hipStreamNonBlocking);	


		diagonal<<<g,b,0,s1>>>(a_d,N,Nm,xi+Nm,yi,zi,n_rec);
		diagonal<<<g,b,0,s2>>>(a_d,N,Nm,xi,yi+Nm,zi,n_rec);
		diagonal<<<g,b,0,s3>>>(a_d,N,Nm,xi,yi,zi-Nm,n_rec);
		//diagonal kernel nuevo 
		return;
	}
	//busca los nuevos puntos a considerar para el siguiente mapeo
	if( tid==0 && Nm!=0){
		//int  Nbloques=(((int) ceil((float)(Nm)/2.0f))+BlockSize-1)/BlockSize;
		int Nbloques=(NmF+BlockSize-1)/BlockSize;
		//printf("nbloques= %i  %i \n", Nbloques, BlockSize);
		dim3 b(BlockSize,BlockSize,BlockSize);
		dim3 g(Nbloques,Nbloques,Nbloques);

		n_rec=n_rec+1;		

 		hipStream_t s1;
		hipStream_t s2;
		hipStream_t s3;

		hipStreamCreateWithFlags(&s1,hipStreamNonBlocking);
		hipStreamCreateWithFlags(&s2,hipStreamNonBlocking);		      
		hipStreamCreateWithFlags(&s3,hipStreamNonBlocking);		
		//
		//int NmF= (int)ceil((float)((Nm)/2.0f));
		//printf("kernel Nm = %i    NmF= %i \n",Nm, NmF);
	
		rec<<<g,b,0,s1>>>(a_d,N,NmF,xi+Nm,yi,zi+NmF,n_rec,N_CORTE,BlockSize);
		rec<<<g,b,0,s2>>>(a_d,N,NmF,xi,yi+Nm,zi+NmF,n_rec,N_CORTE,BlockSize);
		rec<<<g,b,0,s3>>>(a_d,N,NmF,xi,yi,zi-NmF,n_rec,N_CORTE,BlockSize);
	}
}
__global__ void diagonal(int *a_d ,const int N ,int Nm ,int xi ,int yi,int zi,int n_rec){
	int x= blockIdx.x*blockDim.x+threadIdx.x+xi;
	int y= blockIdx.y*blockDim.y+threadIdx.y+yi;
	int z=  blockIdx.z*blockDim.z+threadIdx.z+zi;
	int tid1= z*N*N+y*N+x;
	if(x < N && y < N && z < N){
		if(x+y<= z){
			a_d[tid1]=1;
		}	
	}

}
__global__ void MetNormal(int *a_d,const int N){
	int x= blockIdx.x*blockDim.x+threadIdx.x;
	int y= blockIdx.y*blockDim.y+threadIdx.y;
	int z= blockIdx.z*blockDim.z+threadIdx.z;
	int ind=z*N*N+y*N+x;
	if( x+y<=z){
		a_d[ind]=1;
	}

}
int PrintCu(const int N, int *a){

	for(int k=0;k<N;k++){
		printf("z= %i \n",k);
			for (int i=0;i<N;i++){
				for(int j=0;j<N;j++){
					if(a[N*N*k+i*N+j] != 9){
					printf("%i ",a[N*N*k+i*N+j]);
			   		 }
				    	else{
						printf("* ");
					    }
					}
				printf("\n");
			 }
			 printf("\n");	
		}
	//printf("Ok\n");
	return 0;
}

int Ver_Resultado(const int N, int *a){
	for(int i=0;i<N;i++){
    	for (int j=0;j<N;j++){
        	for(int k=0;k<N;k++){
         		 if(  !((i+j<=k && a[N*N*k+i*N+j]==1) || (i+j>k && a[N*N*k+i*N+j]==9) ) ){
       	  			printf("Error en Matriz \n" );
            	  	exit(1);
              		
            	  }
    		}
    	}
	}
	    //printf("Matriz correcta \n");
        return (0);
}
int main(int argc ,char **argv){
	
	if (argc !=6){
		fprintf(stderr,"error, ejecutar  programa como ./prog N met rep ncorte  BlockSize\n");
		exit(EXIT_FAILURE);
	}
	unsigned long N=atoi(argv[1]);
	unsigned long met=atoi(argv[2]);
	unsigned long rep=atoi(argv[3]);
	unsigned long ncort=atoi(argv[4]);
	//int  nt=atoi(argv[5]);//numero de threads por bloque(eliminar )
	int BSize=atoi(argv[5]);
	int *a,*a_d, xi=0,yi=0,zi=(int) ceil((float)(N/2.0f));
	//double *datos;

	//printf("malloc ...");
	fflush(stdout);
	a=(int*)malloc(sizeof(int)*N*N*N);
	//datos=(double*)malloc(sizeof(double)*rep);

	//printf("ok ...\ncuda malloc...");
	fflush(stdout);
	hipMalloc((void ** ) &a_d,N*N*N*sizeof(int));
	//printf("ok ...\n");
	fflush(stdout);
	
	dim3 Bloque(BSize,BSize,BSize);//un  bloquede nt 

	float NB=(float)N/(float)(2*BSize);
	int B=(int) ceil(NB);
	dim3 Grid(B,B,B);//bgrid  de B*b*b bloque
	dim3 GridBruto((N+BSize-1)/BSize,(N+BSize-1)/BSize,(N+BSize-1)/BSize);
	//printf("inicializando con N= %i ...",N);
	fflush(stdout);
	for(int i=0;i<N;i++){
		for (int j=0;j<N;j++){
			for(int k=0;k<N;k++){
				a[N*N*k+i*N+j]=9;
			}
		}
	}
	//printf(" ok..\n");
	fflush(stdout);
	int n_rec=0;	
	double t1=omp_get_wtime();
	hipMemcpy(a_d,a,N*N*N*sizeof(int),hipMemcpyHostToDevice);
	//printf("calculo GPU...\n");
	fflush(stdout);
	double t2;
	double t3;
	if(ncort >= BSize && (met==1 || met==2 )){
		if(met==1){// aqui se supone que viene un while o for para las iteraciones 
			//printf("Metodo recursivo......\n"); fflush(stdout);
			for(int i=0;i<150;i++){
				rec<<<Grid,Bloque>>>(a_d,N,(int) ceil((float)(N)/2.0f),xi,yi,zi,n_rec,ncort,BSize);
 				hipDeviceSynchronize();	
 				hipError_t error = hipGetLastError();
  				if(error != hipSuccess)
  				{
    // print the CUDA error message and exit
    			printf("CUDA error: %s\n", hipGetErrorString(error));
    			exit(-1);
  }
			}
			t2=omp_get_wtime();
			for(int i=0;i<rep;i++){
				rec<<<Grid,Bloque>>>(a_d,N,(int) ceil((float)(N)/2.0f),xi,yi,zi,n_rec,ncort,BSize);
 				hipDeviceSynchronize();	
 				hipError_t error = hipGetLastError();
  				if(error != hipSuccess)
  				{
  				  // print the CUDA error message and exit
   				 printf("CUDA error: %s\n", hipGetErrorString(error));
    			exit(-1);
 				 }
			}
			t3=omp_get_wtime();
			//printf("ok\n"); 
			fflush(stdout);
		}
		if(met==2){
			//printf("Metodo bruto...\n");
			for(int i =0;i<150;i++){
				MetNormal<<<GridBruto,Bloque>>>(a_d,N);	
				hipDeviceSynchronize();
				}
			t2=omp_get_wtime();
			for(int i =0;i<rep;i++){
				MetNormal<<<GridBruto,Bloque>>>(a_d,N);	
				hipDeviceSynchronize();
				}
			t3=omp_get_wtime();
			fflush(stdout);
			}
		
	}
	else{
		printf("Error, N de corte menor a tamaño de bloque  o metodo invalido\n");
		return;
	}
	//aqui calculo el promedio de los tiempos
	double media=(t3-t2)/rep;
	//printf("Tiempo promedio con %i iteraciones: %f \n",rep,media);
	fflush(stdout);

	hipDeviceSynchronize();
	//printf("ok..\n");
	hipMemcpy(a,a_d,N*N*N*sizeof(int),hipMemcpyDeviceToHost);
	
	double t4=omp_get_wtime();
	/*printf("calculo cpu...");
	fflush(stdout);
	double t5=omp_get_wtime();
	printf("ok..\n");
	printf("verificando...\n");
    */
   	 
	//if(N < 128){
		//PrintCu(N,a);//imprime cubo
   	//}
		Ver_Resultado(N,a);
	//printf("grid : %i %i %i,Bloque:  %i  %i  %i \n",Bloque.x,Bloque.y,Bloque.z ,Grid.x,Grid.y,Grid.z);
	//printf("gridBruto : %i %i %i,Bloque:  %i  %i  %i \n",GridBruto.x,GridBruto.y,GridBruto.z ,Grid.x,Grid.y,Grid.z);
	//printf("tiempo copy a gpu : %f\ntiempo kernel: %f\ntiempo copy to host: %f tiempo total: %f\n",t2-t1,media,t4-t3,t4-t1);
	//printf("tiempo cpu %f\n",t5-t4);
	printf("%f\n",1000*media);
	return 0;
	}

